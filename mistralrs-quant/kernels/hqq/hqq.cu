#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <vector>
#include "cuda_utils.cuh"

inline  unsigned int cdiv(unsigned int a, unsigned int b) { return (a + b - 1) / b;}
#define BLOCK_SIZE 256 //~256 
#define SHARED_SIZE 512 //~512

/*******************************************************************************************************************************************/
/************* 8-bit *************/
/*******************************************************************************************************************************************/
//Simple
extern "C" __global__ void dequantize_8bit_u8_kernel_f32(unsigned char* Wq_packed, float* scale, float* zero, float* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j   = i % w;
	W_r[i]  = ((float)(Wq_packed[i]) - zero[j])*scale[j];  
}

#if __CUDA_ARCH__ >= 630
extern "C" __global__ void dequantize_8bit_u8_kernel_f16(unsigned char* Wq_packed, __half* scale, __half* zero, __half* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j   = i % w;
	W_r[i]  = ((__half)(Wq_packed[i]) - zero[j])*scale[j];  
}
#endif

#if __CUDA_ARCH__ >= 800
extern "C" __global__ void dequantize_8bit_u8_kernel_bf16(unsigned char* Wq_packed, __hip_bfloat16* scale, __hip_bfloat16* zero, __hip_bfloat16* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j   = i % w;
	W_r[i]  = ((__hip_bfloat16)(Wq_packed[i]) - zero[j])*scale[j];  
}
#endif


/*******************************************************************************************************************************************/
/************* 4-bit *************/
/*******************************************************************************************************************************************/

//Simple
/*__global__ void unpack_4bit_u8_kernel(unsigned char* Wq_packed, unsigned char* Wq_unpacked, int n) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i>=n) return;

	Wq_unpacked[i]     = (Wq_packed[i] & 0xF0) >> 4;  //First chunk
	Wq_unpacked[i + n] = (Wq_packed[i] & 0x0F);       //Second chunk	
}*/

//Simple
extern "C"  __global__ void dequantize_4bit_u8_kernel_f32(unsigned char* Wq_packed, float* scale, float* zero, float* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j      = i % w;
	W_r[i]     = ((float)((Wq_packed[i] & 0xF0) >> 4) - zero[j])*scale[j];  //First chunk
	W_r[i + n] = ((float)((Wq_packed[i] & 0x0F))      - zero[j])*scale[j];  //Second chunk
}

#if __CUDA_ARCH__ >= 630
extern "C"  __global__ void dequantize_4bit_u8_kernel_f16(unsigned char* Wq_packed, __half* scale, __half* zero, __half* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j      = i % w;
	W_r[i]     = ((__half)((Wq_packed[i] & 0xF0) >> 4) - zero[j])*scale[j];  //First chunk
	W_r[i + n] = ((__half)((Wq_packed[i] & 0x0F))      - zero[j])*scale[j];  //Second chunk
}
#endif

#if __CUDA_ARCH__ >= 800
extern "C"  __global__ void dequantize_4bit_u8_kernel_bf16(unsigned char* Wq_packed, __hip_bfloat16* scale, __hip_bfloat16* zero, __hip_bfloat16* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j      = i % w;
	W_r[i]     = ((__hip_bfloat16)((Wq_packed[i] & 0xF0) >> 4) - zero[j])*scale[j];  //First chunk
	W_r[i + n] = ((__hip_bfloat16)((Wq_packed[i] & 0x0F))      - zero[j])*scale[j];  //Second chunk
}
#endif

/*******************************************************************************************************************************************/
/************* 2-bit *************/
/*******************************************************************************************************************************************/

//Simple
/*__global__ void unpack_2bit_u8_kernel(unsigned char* Wq_packed, unsigned char* Wq_unpacked, int n) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i>=n) return;

	Wq_unpacked[i]       = (Wq_packed[i] & 0xC0) >> 6;  //1st chunk
	Wq_unpacked[i + n]   = (Wq_packed[i] & 0x30) >> 4;  //2nd chunk
	Wq_unpacked[i + n*2] = (Wq_packed[i] & 0x0C) >> 2;  //3rd chunk	
	Wq_unpacked[i + n*3] = (Wq_packed[i] & 0x03);       //4th chunk	
}*/


//Simple
extern "C" __global__ void dequantize_2bit_u8_kernel_f32(unsigned char* Wq_packed, float* scale, float* zero, float* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j        = i % w;
	W_r[i]       = ((float)((Wq_packed[i] & 0xC0) >> 6) - zero[j])*scale[j];  //1st chunk
	W_r[i + n]   = ((float)((Wq_packed[i] & 0x30) >> 4) - zero[j])*scale[j];  //2nd chunk
	W_r[i + n*2] = ((float)((Wq_packed[i] & 0x0C) >> 2) - zero[j])*scale[j];  //3rd chunk	
	W_r[i + n*3] = ((float)((Wq_packed[i] & 0x03))      - zero[j])*scale[j];  //4th chunk	
}

#if __CUDA_ARCH__ >= 630
extern "C" __global__ void dequantize_2bit_u8_kernel_f16(unsigned char* Wq_packed, __half* scale, __half* zero, __half* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j        = i % w;
	W_r[i]       = ((__half)((Wq_packed[i] & 0xC0) >> 6) - zero[j])*scale[j];  //1st chunk
	W_r[i + n]   = ((__half)((Wq_packed[i] & 0x30) >> 4) - zero[j])*scale[j];  //2nd chunk
	W_r[i + n*2] = ((__half)((Wq_packed[i] & 0x0C) >> 2) - zero[j])*scale[j];  //3rd chunk	
	W_r[i + n*3] = ((__half)((Wq_packed[i] & 0x03))      - zero[j])*scale[j];  //4th chunk	
}
#endif

#if __CUDA_ARCH__ >= 800
extern "C" __global__ void dequantize_2bit_u8_kernel_bf16(unsigned char* Wq_packed, __hip_bfloat16* scale, __hip_bfloat16* zero, __hip_bfloat16* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j        = i % w;
	W_r[i]       = ((__hip_bfloat16)((Wq_packed[i] & 0xC0) >> 6) - zero[j])*scale[j];  //1st chunk
	W_r[i + n]   = ((__hip_bfloat16)((Wq_packed[i] & 0x30) >> 4) - zero[j])*scale[j];  //2nd chunk
	W_r[i + n*2] = ((__hip_bfloat16)((Wq_packed[i] & 0x0C) >> 2) - zero[j])*scale[j];  //3rd chunk	
	W_r[i + n*3] = ((__hip_bfloat16)((Wq_packed[i] & 0x03))      - zero[j])*scale[j];  //4th chunk	
}
#endif


// //Shared
// template <typename scalar_t>
// __global__ void dequantize_2bit_u8_kernel(unsigned char* Wq_packed, scalar_t* scale, scalar_t* zero, scalar_t* W_r, int h, int w) { 
// 	int i = blockIdx.x*blockDim.x + threadIdx.x;
// 	int n = h*w;
// 	int s = threadIdx.x;

// 	if(i>=n) return;

// 	__shared__ unsigned char shared[BLOCK_SIZE];
// 	__shared__ scalar_t shared_meta[BLOCK_SIZE][2];
	
// 	int j             = i % w;
// 	shared[s]         = Wq_packed[i];
// 	shared_meta[s][0] = zero[j];
// 	shared_meta[s][1] = scale[j];
// 	__syncthreads();


// 	W_r[i]       = (scalar_t((shared[s] & 0xC0) >> 6) - shared_meta[s][0])*shared_meta[s][1];  //1st chunk
// 	W_r[i + n]   = (scalar_t((shared[s] & 0x30) >> 4) - shared_meta[s][0])*shared_meta[s][1];  //2nd chunk
// 	W_r[i + n*2] = (scalar_t((shared[s] & 0x0C) >> 2) - shared_meta[s][0])*shared_meta[s][1];  //3rd chunk	
// 	W_r[i + n*3] = (scalar_t((shared[s] & 0x03))      - shared_meta[s][0])*shared_meta[s][1];  //4th chunk	
// }



/*******************************************************************************************************************************************/
/************* 1-bit *************/
/*******************************************************************************************************************************************/

//Simple
/*__global__ void unpack_1bit_u8_kernel(unsigned char* Wq_packed, unsigned char* Wq_unpacked, int n) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i>=n) return;

	Wq_unpacked[i]       = (Wq_packed[i] & 0x80) >> 7;  //1st chunk
	Wq_unpacked[i + n]   = (Wq_packed[i] & 0x40) >> 6;  //2nd chunk
	Wq_unpacked[i + n*2] = (Wq_packed[i] & 0x20) >> 5;  //3rd chunk	
	Wq_unpacked[i + n*3] = (Wq_packed[i] & 0x10) >> 4;  //4th chunk	
	Wq_unpacked[i + n*4] = (Wq_packed[i] & 0x08) >> 3;  //5th chunk	
	Wq_unpacked[i + n*5] = (Wq_packed[i] & 0x04) >> 2;  //6th chunk	
	Wq_unpacked[i + n*6] = (Wq_packed[i] & 0x02) >> 1;  //7th chunk	
	Wq_unpacked[i + n*7] = (Wq_packed[i] & 0x01);       //8th chunk	
}*/

//Simple
extern "C" __global__ void dequantize_1bit_u8_kernel_f32(unsigned char* Wq_packed, float* scale, float* zero, float* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j        = i % w;
	W_r[i]       = ((float)((Wq_packed[i] & 0x80) >> 7) - zero[j])*scale[j];  //1st chunk
	W_r[i + n]   = ((float)((Wq_packed[i] & 0x40) >> 6) - zero[j])*scale[j];  //2nd chunk
	W_r[i + n*2] = ((float)((Wq_packed[i] & 0x20) >> 5) - zero[j])*scale[j];  //3rd chunk	
	W_r[i + n*3] = ((float)((Wq_packed[i] & 0x10) >> 4) - zero[j])*scale[j];  //4th chunk	
	W_r[i + n*4] = ((float)((Wq_packed[i] & 0x08) >> 3) - zero[j])*scale[j];  //5th chunk	
	W_r[i + n*5] = ((float)((Wq_packed[i] & 0x04) >> 2) - zero[j])*scale[j];  //6th chunk	
	W_r[i + n*6] = ((float)((Wq_packed[i] & 0x02) >> 1) - zero[j])*scale[j];  //7th chunk	
	W_r[i + n*7] = ((float)((Wq_packed[i] & 0x01))      - zero[j])*scale[j];  //8th chunk	
}

#if __CUDA_ARCH__ >= 630
extern "C" __global__ void dequantize_1bit_u8_kernel_f16(unsigned char* Wq_packed, __half* scale, __half* zero, __half* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j        = i % w;
	W_r[i]       = ((__half)((Wq_packed[i] & 0x80) >> 7) - zero[j])*scale[j];  //1st chunk
	W_r[i + n]   = ((__half)((Wq_packed[i] & 0x40) >> 6) - zero[j])*scale[j];  //2nd chunk
	W_r[i + n*2] = ((__half)((Wq_packed[i] & 0x20) >> 5) - zero[j])*scale[j];  //3rd chunk	
	W_r[i + n*3] = ((__half)((Wq_packed[i] & 0x10) >> 4) - zero[j])*scale[j];  //4th chunk	
	W_r[i + n*4] = ((__half)((Wq_packed[i] & 0x08) >> 3) - zero[j])*scale[j];  //5th chunk	
	W_r[i + n*5] = ((__half)((Wq_packed[i] & 0x04) >> 2) - zero[j])*scale[j];  //6th chunk	
	W_r[i + n*6] = ((__half)((Wq_packed[i] & 0x02) >> 1) - zero[j])*scale[j];  //7th chunk	
	W_r[i + n*7] = ((__half)((Wq_packed[i] & 0x01))      - zero[j])*scale[j];  //8th chunk	
}
#endif

#if __CUDA_ARCH__ >= 800
extern "C" __global__ void dequantize_1bit_u8_kernel_bf16(unsigned char* Wq_packed, __hip_bfloat16* scale, __hip_bfloat16* zero, __hip_bfloat16* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j        = i % w;
	W_r[i]       = ((__hip_bfloat16)((Wq_packed[i] & 0x80) >> 7) - zero[j])*scale[j];  //1st chunk
	W_r[i + n]   = ((__hip_bfloat16)((Wq_packed[i] & 0x40) >> 6) - zero[j])*scale[j];  //2nd chunk
	W_r[i + n*2] = ((__hip_bfloat16)((Wq_packed[i] & 0x20) >> 5) - zero[j])*scale[j];  //3rd chunk	
	W_r[i + n*3] = ((__hip_bfloat16)((Wq_packed[i] & 0x10) >> 4) - zero[j])*scale[j];  //4th chunk	
	W_r[i + n*4] = ((__hip_bfloat16)((Wq_packed[i] & 0x08) >> 3) - zero[j])*scale[j];  //5th chunk	
	W_r[i + n*5] = ((__hip_bfloat16)((Wq_packed[i] & 0x04) >> 2) - zero[j])*scale[j];  //6th chunk	
	W_r[i + n*6] = ((__hip_bfloat16)((Wq_packed[i] & 0x02) >> 1) - zero[j])*scale[j];  //7th chunk	
	W_r[i + n*7] = ((__hip_bfloat16)((Wq_packed[i] & 0x01))      - zero[j])*scale[j];  //8th chunk	
}
#endif

// //Shared
// template <typename scalar_t>
// __global__ void dequantize_1bit_u8_kernel(unsigned char* Wq_packed, scalar_t* scale, scalar_t* zero, scalar_t* W_r, int h, int w) { 
// 	int i = blockIdx.x*blockDim.x + threadIdx.x;
// 	int s = threadIdx.x;
// 	int n = h*w;
// 	if(i>=n) return;

// 	__shared__ unsigned char shared[BLOCK_SIZE];
// 	__shared__ scalar_t shared_meta[BLOCK_SIZE][2];
	
// 	int j             = i % w;
// 	shared[s]         = Wq_packed[i];
// 	shared_meta[s][0] = zero[j];
// 	shared_meta[s][1] = scale[j];
// 	__syncthreads();

// 	W_r[i]       = (scalar_t((shared[s] & 0x80) >> 7) - shared_meta[s][0])*shared_meta[s][1]; //1st chunk
// 	W_r[i + n]   = (scalar_t((shared[s] & 0x40) >> 6) - shared_meta[s][0])*shared_meta[s][1]; //2nd chunk
// 	W_r[i + n*2] = (scalar_t((shared[s] & 0x20) >> 5) - shared_meta[s][0])*shared_meta[s][1]; //3rd chunk	
// 	W_r[i + n*3] = (scalar_t((shared[s] & 0x10) >> 4) - shared_meta[s][0])*shared_meta[s][1]; //4th chunk	
// 	W_r[i + n*4] = (scalar_t((shared[s] & 0x08) >> 3) - shared_meta[s][0])*shared_meta[s][1]; //5th chunk	
// 	W_r[i + n*5] = (scalar_t((shared[s] & 0x04) >> 2) - shared_meta[s][0])*shared_meta[s][1]; //6th chunk	
// 	W_r[i + n*6] = (scalar_t((shared[s] & 0x02) >> 1) - shared_meta[s][0])*shared_meta[s][1]; //7th chunk	
// 	W_r[i + n*7] = (scalar_t((shared[s] & 0x01))      - shared_meta[s][0])*shared_meta[s][1]; //8th chunk	
// }


/*******************************************************************************************************************************************/
/************* 3-bit *************/
/*******************************************************************************************************************************************/

//Simple
/*__global__ void unpack_3bit_32_kernel(int32_t* Wq_packed, unsigned char* Wq_unpacked, int n) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	if(i>=n) return;

	Wq_unpacked[i]       = (Wq_packed[i] & 0x38000000) >> 27;  //1st chunk
	Wq_unpacked[i + n]   = (Wq_packed[i] & 0x07000000) >> 24;  //2nd chunk
	Wq_unpacked[i + n*2] = (Wq_packed[i] & 0x00E00000) >> 21;  //3rd chunk	
	Wq_unpacked[i + n*3] = (Wq_packed[i] & 0x001C0000) >> 18;  //4th chunk	
	Wq_unpacked[i + n*4] = (Wq_packed[i] & 0x00038000) >> 15;  //5th chunk	
	Wq_unpacked[i + n*5] = (Wq_packed[i] & 0x00007000) >> 12;  //6th chunk	
	Wq_unpacked[i + n*6] = (Wq_packed[i] & 0x00000E00) >> 9;   //7th chunk	
	Wq_unpacked[i + n*7] = (Wq_packed[i] & 0x000001C0) >> 6;   //8th chunk	
	Wq_unpacked[i + n*8] = (Wq_packed[i] & 0x00000038) >> 3;   //9th chunk	
	Wq_unpacked[i + n*9] = (Wq_packed[i] & 0x00000007);        //10th chunk	
}*/


//Simple
extern "C" __global__ void dequantize_3bit_32_kernel_f32(int32_t* Wq_packed, float* scale, float* zero, float* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j        = i % w;
	W_r[i]       = ((float)((Wq_packed[i] & 0x38000000) >> 27) - zero[j])*scale[j];  //1st chunk
	W_r[i + n]   = ((float)((Wq_packed[i] & 0x07000000) >> 24) - zero[j])*scale[j];  //2nd chunk
	W_r[i + n*2] = ((float)((Wq_packed[i] & 0x00E00000) >> 21) - zero[j])*scale[j];  //3rd chunk	
	W_r[i + n*3] = ((float)((Wq_packed[i] & 0x001C0000) >> 18) - zero[j])*scale[j];  //4th chunk	
	W_r[i + n*4] = ((float)((Wq_packed[i] & 0x00038000) >> 15) - zero[j])*scale[j];  //5th chunk	
	W_r[i + n*5] = ((float)((Wq_packed[i] & 0x00007000) >> 12) - zero[j])*scale[j];  //6th chunk	
	W_r[i + n*6] = ((float)((Wq_packed[i] & 0x00000E00) >> 9)  - zero[j])*scale[j];  //7th chunk	
	W_r[i + n*7] = ((float)((Wq_packed[i] & 0x000001C0) >> 6)  - zero[j])*scale[j];  //8th chunk	
	W_r[i + n*8] = ((float)((Wq_packed[i] & 0x00000038) >> 3)  - zero[j])*scale[j];  //9th chunk	
	W_r[i + n*9] = ((float)((Wq_packed[i] & 0x00000007))       - zero[j])*scale[j];  //10th chunk	
}

#if __CUDA_ARCH__ >= 630
extern "C" __global__ void dequantize_3bit_32_kernel_f16(int32_t* Wq_packed, __half* scale, __half* zero, __half* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j        = i % w;
	W_r[i]       = ((__half)((Wq_packed[i] & 0x38000000) >> 27) - zero[j])*scale[j];  //1st chunk
	W_r[i + n]   = ((__half)((Wq_packed[i] & 0x07000000) >> 24) - zero[j])*scale[j];  //2nd chunk
	W_r[i + n*2] = ((__half)((Wq_packed[i] & 0x00E00000) >> 21) - zero[j])*scale[j];  //3rd chunk	
	W_r[i + n*3] = ((__half)((Wq_packed[i] & 0x001C0000) >> 18) - zero[j])*scale[j];  //4th chunk	
	W_r[i + n*4] = ((__half)((Wq_packed[i] & 0x00038000) >> 15) - zero[j])*scale[j];  //5th chunk	
	W_r[i + n*5] = ((__half)((Wq_packed[i] & 0x00007000) >> 12) - zero[j])*scale[j];  //6th chunk	
	W_r[i + n*6] = ((__half)((Wq_packed[i] & 0x00000E00) >> 9)  - zero[j])*scale[j];  //7th chunk	
	W_r[i + n*7] = ((__half)((Wq_packed[i] & 0x000001C0) >> 6)  - zero[j])*scale[j];  //8th chunk	
	W_r[i + n*8] = ((__half)((Wq_packed[i] & 0x00000038) >> 3)  - zero[j])*scale[j];  //9th chunk	
	W_r[i + n*9] = ((__half)((Wq_packed[i] & 0x00000007))       - zero[j])*scale[j];  //10th chunk	
}
#endif

#if __CUDA_ARCH__ >= 800
extern "C" __global__ void dequantize_3bit_32_kernel_bf16(int32_t* Wq_packed, __hip_bfloat16* scale, __hip_bfloat16* zero, __hip_bfloat16* W_r, int h, int w) { 
	int i = blockIdx.x*blockDim.x + threadIdx.x;
	int n = h*w;
	if(i>=n) return;

	int j        = i % w;
	W_r[i]       = ((__hip_bfloat16)((Wq_packed[i] & 0x38000000) >> 27) - zero[j])*scale[j];  //1st chunk
	W_r[i + n]   = ((__hip_bfloat16)((Wq_packed[i] & 0x07000000) >> 24) - zero[j])*scale[j];  //2nd chunk
	W_r[i + n*2] = ((__hip_bfloat16)((Wq_packed[i] & 0x00E00000) >> 21) - zero[j])*scale[j];  //3rd chunk	
	W_r[i + n*3] = ((__hip_bfloat16)((Wq_packed[i] & 0x001C0000) >> 18) - zero[j])*scale[j];  //4th chunk	
	W_r[i + n*4] = ((__hip_bfloat16)((Wq_packed[i] & 0x00038000) >> 15) - zero[j])*scale[j];  //5th chunk	
	W_r[i + n*5] = ((__hip_bfloat16)((Wq_packed[i] & 0x00007000) >> 12) - zero[j])*scale[j];  //6th chunk	
	W_r[i + n*6] = ((__hip_bfloat16)((Wq_packed[i] & 0x00000E00) >> 9)  - zero[j])*scale[j];  //7th chunk	
	W_r[i + n*7] = ((__hip_bfloat16)((Wq_packed[i] & 0x000001C0) >> 6)  - zero[j])*scale[j];  //8th chunk	
	W_r[i + n*8] = ((__hip_bfloat16)((Wq_packed[i] & 0x00000038) >> 3)  - zero[j])*scale[j];  //9th chunk	
	W_r[i + n*9] = ((__hip_bfloat16)((Wq_packed[i] & 0x00000007))       - zero[j])*scale[j];  //10th chunk	
}
#endif
